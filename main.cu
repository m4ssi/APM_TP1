
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <stdlib.h>

__global__ void kernel ( double * a, double * b, double * c, int N)
{
  int i = blockIdx.x * blockDim.x + threadIdx.x;
  if ( i < N)
    {
      c[i] = a[i] + b[i];
    }
}

int main ( int argc, char ** argv)
{
  int N = (argc < 2) ? 1000 : atoi(argv[1]);
  int NN = N*N;
  int size_n = N*sizeof(double);
  double *h_a, *h_b, *h_c;
  double *d_a, *d_b, *d_c;

  h_a = (double *) malloc ( size_n);
  h_b = (double *) malloc ( size_n);
  h_c = (double *) malloc ( size_n);

  // Init values
  for ( int i = 0; i < N; i++)
    {
      h_a[i] = 1;
      h_b[i] = 1;
      h_c[i] = 0;
    }
  
  // Q.4 : Allouer 3 vecteurs de tailles NxN sur le GPU
  hipMalloc ((void **) &d_a, size_n);
  hipMalloc ((void **) &d_b, size_n);
  hipMalloc ((void **) &d_c, size_n);

  hipMemcpy ( d_a, h_a, size_n, hipMemcpyHostToDevice);
  hipMemcpy ( d_b, h_b, size_n, hipMemcpyHostToDevice);

  dim3 dimBlock ( 64, 1, 1);
  dim3 dimGrid ( (N+ dimBlock.x -1)/dimBlock.x, 1, 1);

  kernel<<<dimGrid, dimBlock>>>(d_a, d_b, d_c, N);

  hipMemcpy ( h_c, d_c, size_n, hipMemcpyDeviceToHost);

  // Free on device
  hipFree(d_a);
  hipFree(d_b);
  hipFree(d_c);

  // Free main memory
  free(h_a);
  free(h_b);
  free(h_c);

  return 0;
}