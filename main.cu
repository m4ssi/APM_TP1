
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <stdlib.h>

__global__ void kernel ( double * a, double * b, double * c, int N)
{
  int g_block_i = blockIdx.y * gridDim.x + blockIdx.x;
  int n_threads = blockDim.x * blockDim.y;
  int g_thread_i = g_block_i * n_threads + (threadIdx.y * blockDim.x + threadIdx.x);
  int g_mat_i = g_thread_i / N;
  int g_mat_j = g_thread_i % N;
  for ( int i = 0; i < N; i++)
  {
//	for ( int j = 0; j < N; j++)
//	   {
		c[g_mat_i * N + g_mat_j] = a[g_mat_i * N + i] * b[ i * N + g_mat_j];
//	   }
  }
}

int main ( int argc, char ** argv)
{
  int N = (argc < 2) ? 64 : atoi(argv[1]);
  int NN = N*N;
  int size_n = NN*sizeof(double);
  double *h_a, *h_b, *h_c;
  double *d_a, *d_b, *d_c;

  h_a = (double *) malloc ( size_n);
  h_b = (double *) malloc ( size_n);
  h_c = (double *) malloc ( size_n);

  // Init values
  for ( int i = 0; i < N; i++)
    {
      h_a[i] = 1;
      h_b[i] = 1;
      h_c[i] = 0;
    }
  
  // Q.4 : Allouer 3 vecteurs de tailles NxN sur le GPU
  hipMalloc ((void **) &d_a, size_n);
  hipMalloc ((void **) &d_b, size_n);
  hipMalloc ((void **) &d_c, size_n);

  hipMemcpy ( d_a, h_a, size_n, hipMemcpyHostToDevice);
  hipMemcpy ( d_b, h_b, size_n, hipMemcpyHostToDevice);

  dim3 dimBlock ( 32, 32);
  dim3 dimGrid ( N/dimBlock.x, N/dimBlock.y);

  kernel<<<dimGrid, dimBlock>>>(d_a, d_b, d_c, N);

  hipMemcpy ( h_c, d_c, size_n, hipMemcpyDeviceToHost);



	for	( int i = 0; i < NN; i++)
		printf ("%lf ", h_c[i]);
	printf ("\n");

  // Free on device
  hipFree(d_a);
  hipFree(d_b);
  hipFree(d_c);

  // Free main memory
  free(h_a);
  free(h_b);
  free(h_c);

  return 0;
}
